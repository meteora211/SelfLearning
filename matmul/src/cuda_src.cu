#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "gpu_optimizer.h"

void cuda_executor(void(*cuda_func)(float *, float *, float *, const int, const int, const int),
                   std::shared_ptr<float[]> lhs,
                   std::shared_ptr<float[]> rhs,
                   std::shared_ptr<float[]> res,
                   int M, int N, int K) {
  size_t lhs_size = M * K * sizeof(float);
  size_t rhs_size = K * N * sizeof(float);
  size_t res_size = M * N * sizeof(float);

  float* lhs_device, *rhs_device, *res_device;
  hipMalloc(&lhs_device, lhs_size);
  hipMalloc(&rhs_device, rhs_size);
  hipMalloc(&res_device, res_size);

  hipMemcpy(lhs_device, lhs.get(), lhs_size, hipMemcpyHostToDevice);
  hipMemcpy(rhs_device, rhs.get(), rhs_size, hipMemcpyHostToDevice);
  hipMemcpy(res_device, res.get(), res_size, hipMemcpyHostToDevice);

  const int BM = 32;
  const int BN = 32;

  dim3 block(BN, BM);
  dim3 grid((N + BN - 1) / BN, (M + BM - 1) / BM);
  cuda_func<<<grid, block>>>(lhs_device, rhs_device, res_device, M, N, K);

  hipMemcpy(res.get(), res_device, res_size, hipMemcpyDeviceToHost);


  hipFree(lhs_device);
  hipFree(rhs_device);
  hipFree(res_device);
}


constexpr int block_size = 32;

__global__ void cuda_basic(
    float* lhs, float* rhs, float* res,
    const int M, const int N, const int K) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < M && j < N) {
    float sum = 0.0;
    for (int k = 0; k < K; k++) {
        sum += lhs[i * K + k] * rhs[k * N + j];
    }
    res[i * N + j] = sum;
  }
}

// From 15-418:
// CUDA threads numbered within block in ROW-major order:
// X = column number; Y = row number;
// - Threads with same value of Y map to single warp
// - Threads with same value of Y map and consecutive values of X map to consecutive positions in single warp
// - When single warp access consecutive memory locations, do block read or write
// - When single warp access separated memory locations, requires gather(read) or scatter(write)
__global__ void cuda_transpose(
    float* lhs, float* rhs, float* res,
  const int M, const int N, const int K) {
  // i and j(matrix index for result) is mapped to block in inverted way.
  // Note that it doesn't influence the correctness of result but only the perfermance.
  // Threads within a warp:
  // Regular:  res[i, j : j + 32] = lhs[i, :] * rhs[:, j : j + 32]
  // Inverted: res[i : i + 32, j] = lhs[i : i + 32, :] * rhs[:, j]
  // Read/Write in regular match the memory order and can use block read/write
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < M && j < N) {
    float sum = 0.0;
    for (int k = 0; k < K; k++) {
        sum += lhs[i * K + k] * rhs[k * N + j];
    }
    res[i * N + j] = sum;
  }
}

void matmul_cuda_naive(std::shared_ptr<float[]> lhs, std::shared_ptr<float[]> rhs, std::shared_ptr<float[]> res, int M, int N, int K) {
  cuda_executor(cuda_basic, rhs, lhs, res, M, N, K);
}

void matmul_cuda_transpose(std::shared_ptr<float[]> lhs, std::shared_ptr<float[]> rhs, std::shared_ptr<float[]> res, int M, int N, int K) {
  cuda_executor(cuda_transpose, rhs, lhs, res, M, N, K);
}
