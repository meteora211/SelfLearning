#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "gpu_optimizer.h"
#include <hipblas.h>
#define OFFSET(row, col, ld) ((row) * (ld) + (col))
#define FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])

double cuda_executor(void(*cuda_func)(float *, float *, float *, const int, const int, const int),
                   std::shared_ptr<float[]> lhs,
                   std::shared_ptr<float[]> rhs,
                   std::shared_ptr<float[]> res,
                   int M, int N, int K,
                   dim3 grid,
                   dim3 block) {
  size_t lhs_size = M * K * sizeof(float);
  size_t rhs_size = K * N * sizeof(float);
  size_t res_size = M * N * sizeof(float);

  float* lhs_device, *rhs_device, *res_device;
  hipMalloc(&lhs_device, lhs_size);
  hipMalloc(&rhs_device, rhs_size);
  hipMalloc(&res_device, res_size);

  hipMemcpy(lhs_device, lhs.get(), lhs_size, hipMemcpyHostToDevice);
  hipMemcpy(rhs_device, rhs.get(), rhs_size, hipMemcpyHostToDevice);
  hipMemcpy(res_device, res.get(), res_size, hipMemcpyHostToDevice);

  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);
  hipEventRecord(start);
  cuda_func<<<grid, block>>>(lhs_device, rhs_device, res_device, M, N, K);

  hipEventRecord(end);
  hipEventSynchronize(end);

  float msec;
  hipEventElapsedTime(&msec, start, end);
  auto gflops = get_matmul_GFLOPS(M, N, K,msec/1000);

  hipMemcpy(res.get(), res_device, res_size, hipMemcpyDeviceToHost);


  hipFree(lhs_device);
  hipFree(rhs_device);
  hipFree(res_device);
  return gflops;
}


__global__ void cuda_basic(
    float* lhs, float* rhs, float* res,
    const int M, const int N, const int K) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < M && j < N) {
    float sum = 0.0;
    for (int k = 0; k < K; k++) {
        sum += lhs[i * K + k] * rhs[k * N + j];
    }
    res[i * N + j] = sum;
  }
}

// From 15-418:
// CUDA threads numbered within block in ROW-major order:
// X = column number; Y = row number;
// - Threads with same value of Y map to single warp
// - Threads with same value of Y map and consecutive values of X map to consecutive positions in single warp
// - When single warp access consecutive memory locations, do block read or write
// - When single warp access separated memory locations, requires gather(read) or scatter(write)
__global__ void cuda_transpose(
    float* lhs, float* rhs, float* res,
  const int M, const int N, const int K) {
  // i and j(matrix index for result) is mapped to block in inverted way.
  // Note that it doesn't influence the correctness of result but only the perfermance.
  // Threads within a warp:
  // Regular:  res[i, j : j + 32] = lhs[i, :] * rhs[:, j : j + 32]
  // Inverted: res[i : i + 32, j] = lhs[i : i + 32, :] * rhs[:, j]
  // Read/Write in regular match the memory order and can use block read/write
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < M && j < N) {
    float sum = 0.0;
    for (int k = 0; k < K; k++) {
        sum += lhs[i * K + k] * rhs[k * N + j];
    }
    res[i * N + j] = sum;
  }
}

__global__ void cuda_block(float* lhs, float* rhs, float* res,
    const int M, const int N, const int K) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int bi = threadIdx.y;
  int bj = threadIdx.x;
  const int block_size = 32;
  __shared__ float block_lhs[block_size * block_size];
  __shared__ float block_rhs[block_size * block_size];

  float sum = 0.0;

  for (int k = 0; k < K; k+=block_size) {
    if ((i < M) && ((bj + k) < K)) {
      block_lhs[bi * block_size + bj] = lhs[i * N + (bj + k)];
    } else {
      block_lhs[bi * block_size + bj] = 0;
    }

    if ((j < N) && ((bi + k) < K)) {
      block_rhs[bi * block_size + bj] = rhs[(bi + k) * K + j];
    } else {
      block_rhs[bi * block_size + bj] = 0;
    }

    __syncthreads();

    for (int bk = 0; bk < block_size; ++bk) {
      sum += block_lhs[bi * block_size + bk] * block_rhs[bk * block_size + bj];
    }

    __syncthreads();
  }

  if (i < M && j < N) {
    res[i * N + j] = sum;
  }
}

__global__ void mySgemmV1Aligned(
    float * __restrict__ a, float * __restrict__ b, float * __restrict__ c,
    const int M, const int N, const int K) {

    const int BM = 128;
    const int BN = 128;
    const int BK = 8;
    const int TM = 8;
    const int TN = 8;

    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int tid = ty * blockDim.x + tx;

    __shared__ float s_a[BM][BK];
    __shared__ float s_b[BK][BN];

    float r_c[TM][TN] = {0.0};

    int load_a_smem_m = tid >> 1;
    int load_a_smem_k = (tid & 1) << 2;
    int load_b_smem_k = tid >> 5;
    int load_b_smem_n = (tid & 31) << 2;

    int load_a_gmem_m = by * BM + load_a_smem_m;
    int load_b_gmem_n = bx * BN + load_b_smem_n;

    for (int bk = 0; bk < (K + BK - 1) / BK; bk++) {
        int load_a_gmem_k = bk * BK + load_a_smem_k;
        int load_a_gmem_addr = OFFSET(load_a_gmem_m, load_a_gmem_k, K);
        FLOAT4(s_a[load_a_smem_m][load_a_smem_k]) = FLOAT4(a[load_a_gmem_addr]);
        int load_b_gmem_k = bk * BK + load_b_smem_k;
        int load_b_gmem_addr = OFFSET(load_b_gmem_k, load_b_gmem_n, N);
        FLOAT4(s_b[load_b_smem_k][load_b_smem_n]) = FLOAT4(b[load_b_gmem_addr]);

        __syncthreads();

        #pragma unroll
        for (int k = 0; k < BK; k++) {
            #pragma unroll
            for (int m = 0; m < TM; m++) {
                #pragma unroll
                for (int n = 0; n < TN; n++) {
                    int comp_a_smem_m = ty * TM + m;
                    int comp_b_smem_n = tx * TN + n;
                    r_c[m][n] += s_a[comp_a_smem_m][k] * s_b[k][comp_b_smem_n];
                }
            }
        }

        __syncthreads();
    }

    #pragma unroll
    for (int i = 0; i < TM; i++) {
        int store_c_gmem_m = by * BM + ty * TM + i;
        #pragma unroll
        for (int j = 0; j < TN; j += 4) {
            int store_c_gmem_n = bx * BN + tx * TN + j;
            int store_c_gmem_addr = OFFSET(store_c_gmem_m, store_c_gmem_n, N);
            FLOAT4(c[store_c_gmem_addr]) = FLOAT4(r_c[i][j]);
        }
    }
}

double matmul_cublas(std::shared_ptr<float[]> lhs, std::shared_ptr<float[]> rhs, std::shared_ptr<float[]> res, int M, int N, int K) {
  size_t lhs_size = M * K * sizeof(float);
  size_t rhs_size = K * N * sizeof(float);
  size_t res_size = M * N * sizeof(float);

  float* lhs_device, *rhs_device, *res_device;
  hipMalloc(&lhs_device, lhs_size);
  hipMalloc(&rhs_device, rhs_size);
  hipMalloc(&res_device, res_size);

  hipMemcpy(lhs_device, lhs.get(), lhs_size, hipMemcpyHostToDevice);
  hipMemcpy(rhs_device, rhs.get(), rhs_size, hipMemcpyHostToDevice);
  hipMemcpy(res_device, res.get(), res_size, hipMemcpyHostToDevice);

  hipblasHandle_t cublas_handle;
  hipblasCreate(&cublas_handle);
  float cublas_alpha = 1.0;
  float cublas_beta = 0;
  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);
  hipEventRecord(start);

  //https://docs.nvidia.com/cuda/cublas/index.html?highlight=hipblasSgemm#cublas-t-gemm
  hipblasSgemm(cublas_handle,
              HIPBLAS_OP_N,
              HIPBLAS_OP_N,
              // TODO: not sure why M/N/K got wrong result
              // M, N, K,
              N, M, K,
              &cublas_alpha,
              rhs_device, N,
              lhs_device, K,
              &cublas_beta,
              res_device, N);

  hipEventRecord(end);
  hipEventSynchronize(end);

  float msec;
  hipEventElapsedTime(&msec, start, end);
  auto gflops = get_matmul_GFLOPS(M, N, K,msec/1000);

  hipMemcpy(res.get(), res_device, res_size, hipMemcpyDeviceToHost);


  hipFree(lhs_device);
  hipFree(rhs_device);
  hipFree(res_device);
  return gflops;
}

double matmul_cuda_naive(std::shared_ptr<float[]> lhs, std::shared_ptr<float[]> rhs, std::shared_ptr<float[]> res, int M, int N, int K) {
  const int BM = 32;
  const int BN = 32;

  dim3 grid((N + BN - 1) / BN, (M + BM - 1) / BM);
  dim3 block(BN, BM);
  return cuda_executor(cuda_basic, rhs, lhs, res, M, N, K, grid, block);
}

double matmul_cuda_transpose(std::shared_ptr<float[]> lhs, std::shared_ptr<float[]> rhs, std::shared_ptr<float[]> res, int M, int N, int K) {
  const int BM = 32;
  const int BN = 32;

  dim3 grid((N + BN - 1) / BN, (M + BM - 1) / BM);
  dim3 block(BN, BM);
  return cuda_executor(cuda_transpose, rhs, lhs, res, M, N, K, grid, block);
}

double matmul_cuda_block(std::shared_ptr<float[]> lhs, std::shared_ptr<float[]> rhs, std::shared_ptr<float[]> res, int M, int N, int K) {
  const int BM = 32;
  const int BN = 32;

  dim3 grid((N + BN - 1) / BN, (M + BM - 1) / BM);
  dim3 block(BN, BM);
  return cuda_executor(cuda_block, rhs, lhs, res, M, N, K, grid, block);
}

/* void matmul_cuda_copied_code(std::shared_ptr<float[]> lhs, std::shared_ptr<float[]> rhs, std::shared_ptr<float[]> res, int M, int N, int K) { */
/*   const int BM = 128, BN = 128, TM = 8, TN = 8; */
/*   dim3 grid((N + BN - 1) / BN, (M + BM - 1) / BM); */
/*   dim3 block(BN / TN, BM / TM); */

/*   cuda_executor(mySgemmV1Aligned, rhs, lhs, res, M, N, K, grid, block); */
/* } */
