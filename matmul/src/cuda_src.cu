#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "gpu_optimizer.h"
#include <hipblas.h>

constexpr int inner_block = 32;

void cuda_executor(void(*cuda_func)(float *, float *, float *, const int, const int, const int),
                   std::shared_ptr<float[]> lhs,
                   std::shared_ptr<float[]> rhs,
                   std::shared_ptr<float[]> res,
                   int M, int N, int K,
                   dim3 block,
                   dim3 grid) {
  size_t lhs_size = M * K * sizeof(float);
  size_t rhs_size = K * N * sizeof(float);
  size_t res_size = M * N * sizeof(float);

  float* lhs_device, *rhs_device, *res_device;
  hipMalloc(&lhs_device, lhs_size);
  hipMalloc(&rhs_device, rhs_size);
  hipMalloc(&res_device, res_size);

  hipMemcpy(lhs_device, lhs.get(), lhs_size, hipMemcpyHostToDevice);
  hipMemcpy(rhs_device, rhs.get(), rhs_size, hipMemcpyHostToDevice);
  hipMemcpy(res_device, res.get(), res_size, hipMemcpyHostToDevice);

  cuda_func<<<grid, block>>>(lhs_device, rhs_device, res_device, M, N, K);

  hipMemcpy(res.get(), res_device, res_size, hipMemcpyDeviceToHost);


  hipFree(lhs_device);
  hipFree(rhs_device);
  hipFree(res_device);
}


__global__ void cuda_basic(
    float* lhs, float* rhs, float* res,
    const int M, const int N, const int K) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < M && j < N) {
    float sum = 0.0;
    for (int k = 0; k < K; k++) {
        sum += lhs[i * K + k] * rhs[k * N + j];
    }
    res[i * N + j] = sum;
  }
}

// From 15-418:
// CUDA threads numbered within block in ROW-major order:
// X = column number; Y = row number;
// - Threads with same value of Y map to single warp
// - Threads with same value of Y map and consecutive values of X map to consecutive positions in single warp
// - When single warp access consecutive memory locations, do block read or write
// - When single warp access separated memory locations, requires gather(read) or scatter(write)
__global__ void cuda_transpose(
    float* lhs, float* rhs, float* res,
  const int M, const int N, const int K) {
  // i and j(matrix index for result) is mapped to block in inverted way.
  // Note that it doesn't influence the correctness of result but only the perfermance.
  // Threads within a warp:
  // Regular:  res[i, j : j + 32] = lhs[i, :] * rhs[:, j : j + 32]
  // Inverted: res[i : i + 32, j] = lhs[i : i + 32, :] * rhs[:, j]
  // Read/Write in regular match the memory order and can use block read/write
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < M && j < N) {
    float sum = 0.0;
    for (int k = 0; k < K; k++) {
        sum += lhs[i * K + k] * rhs[k * N + j];
    }
    res[i * N + j] = sum;
  }
}

/* __global__ void cuda_block(float* lhs, float* rhs, float* res, */
/*     const int M, const int N, const int K) { */
/*   int i = blockIdx.y * blockDim.y + threadIdx.y; */
/*   int j = blockIdx.x * blockDim.x + threadIdx.x; */
/*   int bi = threadIdx.y; */
/*   int bj = threadIdx.x; */
/*   const int BM = 256; */
/*   const int BN = 256; */
/*   const int BK = 32; */
/*   __shared__ float block_lhs[BM * BK]; */
/*   __shared__ float block_rhs[BK * BN]; */

/*   float block_res[inner_block_size * inner_block_size] = {0}; */


/*   float sum = 0.0; */
/*   for (int k = 0; k < K; k+=BK) { */
/*     block_lhs[bi * inner_block_size + bj] = lhs[(i + )] */
/*     block_rhs[bj * inner_block_size + bi] = rhs[i, k] */

/*     sum += lhs[i * K + k] * rhs[k * N + j]; */
/*   } */
/*   if (i < M && j < N) { */
/*     res[i * N + j] = sum; */
/*   } */
/* } */

void matmul_cublas(std::shared_ptr<float[]> lhs, std::shared_ptr<float[]> rhs, std::shared_ptr<float[]> res, int M, int N, int K) {
  size_t lhs_size = M * K * sizeof(float);
  size_t rhs_size = K * N * sizeof(float);
  size_t res_size = M * N * sizeof(float);

  float* lhs_device, *rhs_device, *res_device;
  hipMalloc(&lhs_device, lhs_size);
  hipMalloc(&rhs_device, rhs_size);
  hipMalloc(&res_device, res_size);

  hipMemcpy(lhs_device, lhs.get(), lhs_size, hipMemcpyHostToDevice);
  hipMemcpy(rhs_device, rhs.get(), rhs_size, hipMemcpyHostToDevice);
  hipMemcpy(res_device, res.get(), res_size, hipMemcpyHostToDevice);

  hipblasHandle_t cublas_handle;
  hipblasCreate(&cublas_handle);
  float cublas_alpha = 1.0;
  float cublas_beta = 0;
  //https://docs.nvidia.com/cuda/cublas/index.html?highlight=hipblasSgemm#cublas-t-gemm
  hipblasSgemm(cublas_handle,
              HIPBLAS_OP_N,
              HIPBLAS_OP_N,
              // TODO: not sure why M/N/K got wrong result
              // M, N, K,
              N, M, K,
              &cublas_alpha,
              rhs_device, N,
              lhs_device, K,
              &cublas_beta,
              res_device, N);

  hipMemcpy(res.get(), res_device, res_size, hipMemcpyDeviceToHost);


  hipFree(lhs_device);
  hipFree(rhs_device);
  hipFree(res_device);
}

void matmul_cuda_naive(std::shared_ptr<float[]> lhs, std::shared_ptr<float[]> rhs, std::shared_ptr<float[]> res, int M, int N, int K) {
  const int BM = 32;
  const int BN = 32;

  dim3 grid((N + BN - 1) / BN, (M + BM - 1) / BM);
  dim3 block(BN, BM);
  cuda_executor(cuda_basic, rhs, lhs, res, M, N, K, grid, block);
}

void matmul_cuda_transpose(std::shared_ptr<float[]> lhs, std::shared_ptr<float[]> rhs, std::shared_ptr<float[]> res, int M, int N, int K) {
  const int BM = 32;
  const int BN = 32;

  dim3 grid((N + BN - 1) / BN, (M + BM - 1) / BM);
  dim3 block(BN, BM);
  cuda_executor(cuda_transpose, rhs, lhs, res, M, N, K, grid, block);
}

/* void matmul_cuda_block(std::shared_ptr<float[]> lhs, std::shared_ptr<float[]> rhs, std::shared_ptr<float[]> res, int M, int N, int K) { */
/*   const int BM = 256; */
/*   const int BN = 256; */

/*   dim3 grid((N + BN - 1) / BN, (M + BM - 1) / BM); */
/*   dim3 block(BN / inner_block, BM / inner_block); */
/*   cuda_executor(cuda_block, rhs, lhs, res, M, N, K, grid, block); */
/* } */
