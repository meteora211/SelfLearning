#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "gpu_optimizer.h"

__global__ void naiveSgemm(
    float* a, float* b, float* c,
    const int M, const int N, const int K) {

    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < M && j < N) {
        float sum = 0.0;
        for (int k = 0; k < K; k++) {
            sum += a[i * K + k] * b[k * N + j];
        }
        c[i * N + j] = sum;
    }
}

void matmul_cuda_naive(std::shared_ptr<float[]> lhs, std::shared_ptr<float[]> rhs, std::shared_ptr<float[]> res, int M, int N, int K) {
  size_t lhs_size = M * K * sizeof(float);
  size_t rhs_size = K * N * sizeof(float);
  size_t res_size = M * N * sizeof(float);

  float* lhs_device, *rhs_device, *res_device;
  hipMalloc(&lhs_device, lhs_size);
  hipMalloc(&rhs_device, rhs_size);
  hipMalloc(&res_device, res_size);

  hipMemcpy(lhs_device, lhs.get(), lhs_size, hipMemcpyHostToDevice);
  hipMemcpy(rhs_device, rhs.get(), rhs_size, hipMemcpyHostToDevice);
  hipMemcpy(res_device, res.get(), res_size, hipMemcpyHostToDevice);

  const int BM = 32;
  const int BN = 32;
  dim3 block(BN, BM);
  dim3 grid((N + BN - 1) / BN, (M + BM - 1) / BM);

  naiveSgemm<<<grid, block>>>(lhs_device, rhs_device, res_device, M, N, K);

  hipMemcpy(res.get(), res_device, res_size, hipMemcpyDeviceToHost);

  hipFree(lhs_device);
  hipFree(rhs_device);
  hipFree(res_device);
}
