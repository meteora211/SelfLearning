#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "gpu_optimizer.h"
#include <hipblas.h>

void cuda_executor(void(*cuda_func)(float *, float *, float *, const int, const int, const int),
                   std::shared_ptr<float[]> lhs,
                   std::shared_ptr<float[]> rhs,
                   std::shared_ptr<float[]> res,
                   int M, int N, int K,
                   dim3 grid,
                   dim3 block) {
  size_t lhs_size = M * K * sizeof(float);
  size_t rhs_size = K * N * sizeof(float);
  size_t res_size = M * N * sizeof(float);

  float* lhs_device, *rhs_device, *res_device;
  hipMalloc(&lhs_device, lhs_size);
  hipMalloc(&rhs_device, rhs_size);
  hipMalloc(&res_device, res_size);

  hipMemcpy(lhs_device, lhs.get(), lhs_size, hipMemcpyHostToDevice);
  hipMemcpy(rhs_device, rhs.get(), rhs_size, hipMemcpyHostToDevice);
  hipMemcpy(res_device, res.get(), res_size, hipMemcpyHostToDevice);

  cuda_func<<<grid, block>>>(lhs_device, rhs_device, res_device, M, N, K);

  hipMemcpy(res.get(), res_device, res_size, hipMemcpyDeviceToHost);


  hipFree(lhs_device);
  hipFree(rhs_device);
  hipFree(res_device);
}


__global__ void cuda_basic(
    float* lhs, float* rhs, float* res,
    const int M, const int N, const int K) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < M && j < N) {
    float sum = 0.0;
    for (int k = 0; k < K; k++) {
        sum += lhs[i * K + k] * rhs[k * N + j];
    }
    res[i * N + j] = sum;
  }
}

// From 15-418:
// CUDA threads numbered within block in ROW-major order:
// X = column number; Y = row number;
// - Threads with same value of Y map to single warp
// - Threads with same value of Y map and consecutive values of X map to consecutive positions in single warp
// - When single warp access consecutive memory locations, do block read or write
// - When single warp access separated memory locations, requires gather(read) or scatter(write)
__global__ void cuda_transpose(
    float* lhs, float* rhs, float* res,
  const int M, const int N, const int K) {
  // i and j(matrix index for result) is mapped to block in inverted way.
  // Note that it doesn't influence the correctness of result but only the perfermance.
  // Threads within a warp:
  // Regular:  res[i, j : j + 32] = lhs[i, :] * rhs[:, j : j + 32]
  // Inverted: res[i : i + 32, j] = lhs[i : i + 32, :] * rhs[:, j]
  // Read/Write in regular match the memory order and can use block read/write
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < M && j < N) {
    float sum = 0.0;
    for (int k = 0; k < K; k++) {
        sum += lhs[i * K + k] * rhs[k * N + j];
    }
    res[i * N + j] = sum;
  }
}

__global__ void cuda_block(float* lhs, float* rhs, float* res,
    const int M, const int N, const int K) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int bi = threadIdx.y;
  int bj = threadIdx.x;
  const int block_size = 32;
  __shared__ float block_lhs[block_size * block_size];
  __shared__ float block_rhs[block_size * block_size];

  float sum = 0.0;

  for (int k = 0; k < K; k+=block_size) {
    if ((i < M) && ((bj + k) < K)) {
      block_lhs[bi * block_size + bj] = lhs[i * N + (bj + k)];
    } else {
      block_lhs[bi * block_size + bj] = 0;
    }

    if ((j < N) && ((bi + k) < K)) {
      block_rhs[bi * block_size + bj] = rhs[(bi + k) * K + j];
    } else {
      block_rhs[bi * block_size + bj] = 0;
    }

    __syncthreads();

    for (int bk = 0; bk < block_size; ++bk) {
      sum += block_lhs[bi * block_size + bk] * block_rhs[bk * block_size + bj];
    }

    __syncthreads();
  }

  if (i < M && j < N) {
    res[i * N + j] = sum;
  }
}

void matmul_cublas(std::shared_ptr<float[]> lhs, std::shared_ptr<float[]> rhs, std::shared_ptr<float[]> res, int M, int N, int K) {
  size_t lhs_size = M * K * sizeof(float);
  size_t rhs_size = K * N * sizeof(float);
  size_t res_size = M * N * sizeof(float);

  float* lhs_device, *rhs_device, *res_device;
  hipMalloc(&lhs_device, lhs_size);
  hipMalloc(&rhs_device, rhs_size);
  hipMalloc(&res_device, res_size);

  hipMemcpy(lhs_device, lhs.get(), lhs_size, hipMemcpyHostToDevice);
  hipMemcpy(rhs_device, rhs.get(), rhs_size, hipMemcpyHostToDevice);
  hipMemcpy(res_device, res.get(), res_size, hipMemcpyHostToDevice);

  hipblasHandle_t cublas_handle;
  hipblasCreate(&cublas_handle);
  float cublas_alpha = 1.0;
  float cublas_beta = 0;
  //https://docs.nvidia.com/cuda/cublas/index.html?highlight=hipblasSgemm#cublas-t-gemm
  hipblasSgemm(cublas_handle,
              HIPBLAS_OP_N,
              HIPBLAS_OP_N,
              // TODO: not sure why M/N/K got wrong result
              // M, N, K,
              N, M, K,
              &cublas_alpha,
              rhs_device, N,
              lhs_device, K,
              &cublas_beta,
              res_device, N);

  hipMemcpy(res.get(), res_device, res_size, hipMemcpyDeviceToHost);


  hipFree(lhs_device);
  hipFree(rhs_device);
  hipFree(res_device);
}

void matmul_cuda_naive(std::shared_ptr<float[]> lhs, std::shared_ptr<float[]> rhs, std::shared_ptr<float[]> res, int M, int N, int K) {
  const int BM = 32;
  const int BN = 32;

  dim3 grid((N + BN - 1) / BN, (M + BM - 1) / BM);
  dim3 block(BN, BM);
  cuda_executor(cuda_basic, rhs, lhs, res, M, N, K, grid, block);
}

void matmul_cuda_transpose(std::shared_ptr<float[]> lhs, std::shared_ptr<float[]> rhs, std::shared_ptr<float[]> res, int M, int N, int K) {
  const int BM = 32;
  const int BN = 32;

  dim3 grid((N + BN - 1) / BN, (M + BM - 1) / BM);
  dim3 block(BN, BM);
  cuda_executor(cuda_transpose, rhs, lhs, res, M, N, K, grid, block);
}

void matmul_cuda_block(std::shared_ptr<float[]> lhs, std::shared_ptr<float[]> rhs, std::shared_ptr<float[]> res, int M, int N, int K) {
  const int BM = 32;
  const int BN = 32;

  dim3 grid((N + BN - 1) / BN, (M + BM - 1) / BM);
  dim3 block(BN, BM);
  cuda_executor(cuda_block, rhs, lhs, res, M, N, K, grid, block);
}
